#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
// #include <cstdlib>
#include <cassert>
#include <random>
// #include <algorithm>
// #include <cstdint>
// #include <complex>
// #include <array>
// #include <vector>
// #include <map>

#include <Eigen/Eigenvalues>



#include "gpu_header.h"
#include "dirac.h"



// gauge
// using G = std::complex<double>; // U(1); abs(u)=1, u \in G
// using GaugeField = std::vector<std::array<G, 5>>; // U[x][mu], U[x][0] = 0.0


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  const Complex I = Complex(0.0, 1.0);

#ifdef _OPENMP
  omp_set_num_threads(12);
#endif

  // int device;
  // CUDA_CHECK(hipGetDeviceCount(&device));
  // hipDeviceProp_t device_prop[device];
  // hipGetDeviceProperties(&device_prop[0], 0);
  // std::cout << "# dev = " << device_prop[0].name << std::endl;
  // CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  // std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  int Ls = 4; // 8
  int seed = 1;
  double width = 0.0;

  if(argc==4){
    Ls = atoi(argv[1]);
    seed = atoi(argv[2]);
    width = atof(argv[3]);
  }
  std::cout << "Ls = " << Ls << std::endl;
  std::cout << "seed = " << seed << std::endl;
  std::cout << "width = " << width << std::endl;

  // std::vector<int> L5{ 5, 3,3,3,16, 2};
  // std::vector<int> L5{ 5, 4,4,4,4, 1};
  // std::vector<int> L5{ 5, 6,6,6,6, 1};
  std::vector<int> L5{ 5, 4,4,4,4, Ls};
  // std::vector<int> L5{ 5, 2,2,2,2, 4};
  // std::vector<int> L5{ 5, 2,2,2,2, 2};
  // for(auto elem : L5) std::cout << elem << std::endl;

  DomainWall Ddw(L5);

  std::mt19937 gen(seed);
  std::normal_distribution d{0.0, width};

  GaugeField u(Ddw.vol);
  for(Idx i=0; i<Ddw.vol; i++){
    for(int mu=1; mu<=4; mu++){
      // u[i][mu] = 1.0;
      u[i][mu] = std::exp( I*d(gen) );
    }
  }

  // Eigen::MatrixXcd HW = Ddw.get_Hw( -Ddw.M5, u );
  Eigen::MatrixXcd mat = Ddw.get_X( -Ddw.M5, u );

  // {
  //   std::ofstream file("real.dat", std::ios::trunc);
  //   file << "# real:" << std::endl;
  //   file << mat.real() << std::endl;
  // }
  // {
  //   std::ofstream file("imag.dat", std::ios::trunc);
  //   file << "# imag:" << std::endl;
  //   file << mat.imag() << std::endl;
  // }


  // =========================================

  const int n = mat.cols(); // Number of rows (or columns) of matrix A.

  // std::vector<std::complex<double>> U(n*n);
  // std::vector<std::complex<double>> VH(n*n);

  // {
  //   // cusolver
  //   hipsolverHandle_t handle = NULL;
  //   hipStream_t stream = NULL;
  //   hipsolverDnParams_t params = NULL;

  //   const int lda = n;

  //   CuC *A;
  //   double *S;
  //   A = (CuC*)malloc(n*n*CD);
  //   S = (double*)malloc(n*DB);
  //   for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(mat(i,j));
  //   for(int i=0; i<n; i++) S[i] = 0.;

  //   CuC *d_A, *d_U, *d_VT;
  //   double *d_S;

  //   signed char jobu = 'A';
  //   signed char jobvt = 'A';
  //   int ldu = n;
  //   int ldvt = n;
  //   //
  //   int info = 0;
  //   int *d_info = nullptr;

  //   size_t workspaceInBytesOnDevice = 0; /* size of workspace */
  //   void *d_work = nullptr;              /* device workspace */
  //   size_t workspaceInBytesOnHost = 0;   /* size of workspace */
  //   void *h_work = nullptr;              /* host workspace for */

  //   /* step 1: create cusolver handle, bind a stream */
  //   CUSOLVER_CHECK(hipsolverDnCreate(&handle));
  //   CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  //   CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
  //   CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  //   CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
  //   CUDA_CHECK(hipMalloc( &d_S, DB * n ));
  //   CUDA_CHECK(hipMalloc( &d_U, CD * n*n ));
  //   CUDA_CHECK(hipMalloc( &d_VT, CD * n*n ));
  //   CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

  //   CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );
  //   CUDA_CHECK( hipMemset(d_S, 0, DB * n) );
  //   CUDA_CHECK( hipMemset(d_U, 0, CD * n*n) );
  //   CUDA_CHECK( hipMemset(d_VT, 0, CD * n*n) );

  //   CUSOLVER_CHECK( cusolverDnXgesvd_bufferSize( handle,
  //                                                params,
  //                                                jobu,
  //                                                jobvt,
  //                                                n,
  //                                                n,
  //                                                HIP_C_64F,
  //                                                d_A, // device
  //                                                lda,
  //                                                HIP_R_64F,
  //                                                d_S, // Array holding the computed eigenvalues of A
  //                                                HIP_C_64F,
  //                                                d_U,
  //                                                ldu,
  //                                                HIP_C_64F,
  //                                                d_VT,
  //                                                ldvt,
  //                                                HIP_C_64F,
  //                                                &workspaceInBytesOnDevice,
  //                                                &workspaceInBytesOnHost)
  //                   );

  //   CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
  //   h_work = malloc(workspaceInBytesOnHost);

  //   // step 4: compute spectrum
  //   CUSOLVER_CHECK( cusolverDnXgesvd( handle,
  //                                     params,
  //                                     jobu,
  //                                     jobvt,
  //                                     n,
  //                                     n,
  //                                     HIP_C_64F,
  //                                     d_A,
  //                                     lda,
  //                                     HIP_R_64F,
  //                                     d_S,
  //                                     HIP_C_64F,
  //                                     d_U,
  //                                     ldu,
  //                                     HIP_C_64F,
  //                                     d_VT,
  //                                     ldvt,
  //                                     HIP_C_64F,
  //                                     d_work, // void *bufferOnDevice,
  //                                     workspaceInBytesOnDevice,
  //                                     h_work, // void *bufferOnHost,
  //                                     workspaceInBytesOnHost,
  //                                     d_info)
  //                   );

  //   // ---------------------------------------------

  //   CUDA_CHECK(hipMemcpy( S, d_S, DB*n, D2H) );
  //   CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

  //   CUDA_CHECK(hipMemcpy( reinterpret_cast<CuC*>(U.data()), d_U, CD * n*n, D2H ));
  //   CUDA_CHECK(hipMemcpy( reinterpret_cast<CuC*>(VH.data()), d_VT, CD * n*n, D2H ));

  //   for(Idx i=0; i<n; i++) std::cout << S[i] << std::endl;

  //   std::cout << "# info (0=success) = " << info << std::endl;
  //   assert( info==0 );

  //   /* free resources */
  //   free(A);
  //   free(S);
  //   free(h_work);

  //   CUDA_CHECK(hipFree(d_A));
  //   CUDA_CHECK(hipFree(d_S));
  //   CUDA_CHECK(hipFree(d_U));
  //   CUDA_CHECK(hipFree(d_VT));
  //   CUDA_CHECK(hipFree(d_info));
  //   CUDA_CHECK(hipFree(d_work));

  //   CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
  //   CUSOLVER_CHECK(hipsolverDnDestroy(handle));
  //   CUDA_CHECK(hipStreamDestroy(stream));
  // }

  // eigen, cusolver: both column major
  // Eigen::MatrixXcd eigenU = Eigen::Map<Eigen::MatrixXcd>(U.data(), n, n);
  // Eigen::MatrixXcd eigenVH = Eigen::Map<Eigen::MatrixXcd>(VH.data(), n, n);

  Eigen::BDCSVD<Eigen::MatrixXcd, Eigen::ComputeFullU | Eigen::ComputeFullV> svd(n,n);
  svd.compute(mat);
  Eigen::MatrixXcd eigenU = svd.matrixU();
  Eigen::MatrixXcd eigenVH = svd.matrixV().adjoint();
  Eigen::MatrixXcd Dov = Eigen::MatrixXcd::Identity(n,n) + eigenU*eigenVH;

  std::cout << "eigenU check: " << ( eigenU * eigenU.adjoint() - Eigen::MatrixXcd::Identity(n,n) ).norm() << std::endl;
  std::cout << "eigenVH check: " << ( eigenVH * eigenVH.adjoint() - Eigen::MatrixXcd::Identity(n,n) ).norm() << std::endl;

  {
    // cusolver
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    // const int n = mat.cols(); // Number of rows (or columns) of matrix A.
    const int lda = n;

    CuC *A, *W;
    A = (CuC*)malloc(n*n*CD);
    W = (CuC*)malloc(n*CD);
    for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(Dov(i,j));
    for(int i=0; i<n; i++) W[i] = cplx(0.);

    CuC *d_A, *d_W, *d_VL, *d_VR;

    hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_NOVECTOR;
    int ldvl = n;
    int ldvr = n;
    //
    int info = 0;
    int *d_info = nullptr;

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for */

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
    CUDA_CHECK(hipMalloc( &d_W, CD * n ));
    CUDA_CHECK(hipMalloc( &d_VL, CD * n*n ));
    CUDA_CHECK(hipMalloc( &d_VR, CD * n*n ));
    CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

    CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );
    CUDA_CHECK( hipMemset(d_W, 0, CD * n) );
    CUDA_CHECK( hipMemset(d_VL, 0, CD * n*n) );
    CUDA_CHECK( hipMemset(d_VR, 0, CD * n*n) );


    CUSOLVER_CHECK( cusolverDnXgeev_bufferSize( handle,
                                                params,
                                                jobvl,
                                                jobvr,
                                                n,
                                                HIP_C_64F,
                                                d_A, // device
                                                lda,
                                                HIP_C_64F,
                                                d_W, // Array holding the computed eigenvalues of A
                                                HIP_C_64F,
                                                d_VL,
                                                ldvl,
                                                HIP_C_64F,
                                                d_VR,
                                                ldvr,
                                                HIP_C_64F,
                                                &workspaceInBytesOnDevice,
                                                &workspaceInBytesOnHost)
                    );

    CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
    h_work = malloc(workspaceInBytesOnHost);

    // step 4: compute spectrum
    CUSOLVER_CHECK( cusolverDnXgeev( handle,
                                     params,
                                     jobvl,
                                     jobvr,
                                     n,
                                     HIP_C_64F,
                                     d_A,
                                     lda,
                                     HIP_C_64F,
                                     d_W,
                                     HIP_C_64F,
                                     d_VL,
                                     ldvl,
                                     HIP_C_64F,
                                     d_VR,
                                     ldvr,
                                     HIP_C_64F,
                                     d_work, // void *bufferOnDevice,
                                     workspaceInBytesOnDevice,
                                     h_work, // void *bufferOnHost,
                                     workspaceInBytesOnHost,
                                     d_info)
                    );

    // ---------------------------------------------

    CUDA_CHECK(hipMemcpy( W, d_W, CD*n, D2H) );
    CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

    // std::vector<std::complex<double>> vr(n*n);
    // for(Idx i=0; i<N; i++) gmfourth(d_VL+i*N, d_VR+i*N);
    // CUDA_CHECK(hipMemcpy( reinterpret_cast<CuC*>(vr.data()), d_VL, CD * n*n, D2H ));

    std::cout << "# info (0=success) = " << info << std::endl;
    assert( info==0 );

    std::vector<double> re(n), im(n);
    for(int i=0; i<n; i++) {
      re[i] = real(W[i]);
      im[i] = imag(W[i]);
    }
    // std::sort(res.begin(), res.end());
    for(int i=0; i<n; i++) std::cout << i << " "
                                     << re[i] << " "
                                     << im[i] << " "
                                     << std::endl;


    // for(int i=0; i<n; i++) std::clog << real(vr[i]) << " " << imag(vr[i]) << std::endl;
    // }
    {
      std::ofstream file("ev_Dov_seed"+std::to_string(seed)+"_width"+std::to_string(width)+".dat", std::ios::trunc);
      file << std::scientific << std::setprecision(15);
      file << "# ev" << std::endl;
      for(int i=0; i<n; i++) file << i << " " << re[i] << " " << im[i] << std::endl;

      std::complex<double> log_total;
      // #ifdef _OPENMP
      // #pragma omp parallel for reduction(+:log_total)
      // #endif
      for(int i=0; i<n; i++) {
        log_total += std::log(re[i]+I*im[i]);
      }
      std::cout << "log_total = " << log_total << std::endl;
    }

    /* free resources */
    free(A);
    free(W);
    free(h_work);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_VL));
    CUDA_CHECK(hipFree(d_VR));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
  }




  // std::cout << "get Hw" << std::endl;
  // const Eigen::MatrixXcd Hw = Ddw.get_Hw( -Ddw.M5 );
  // std::cout << "calc SVD" << std::endl;
  // Eigen::BDCSVD<Eigen::MatrixXcd, Eigen::ComputeThinU | Eigen::ComputeThinV> solver(mat);

  // std::cout << "calc V" << std::endl;
  // const Eigen::MatrixXcd V = solver.matrixU() * solver.matrixV().adjoint();
  // const Eigen::MatrixXcd Dov = Eigen::MatrixXcd::Identity(4*Ddw.vol, 4*Ddw.vol) + V;
  // std::cout << "calc det" << std::endl;
  // std::cout << std::log(Dov.determinant()) << std::endl;



  return 0; // EXIT_SUCCESS;
}

